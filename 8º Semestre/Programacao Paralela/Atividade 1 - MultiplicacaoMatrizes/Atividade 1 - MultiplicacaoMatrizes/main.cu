#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <>


// Fun��o para multiplica��o de matrizes em formato serial
void serialMatrixMultiplication(int* matrix, int* auxiliarMatrix, int* resultMatrix, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int sum = 0;
            for (int k = 0; k < n; ++k) {
                sum += matrix[i * n + k] * auxiliarMatrix[k * n + j];
            }
            resultMatrix[i * n + j] = sum;
        }
    }
}

// Kernel para multiplica��o de matrizes em paralelo
__global__ void parallelMatrixMultiplication(int* matrix1, int* matrix2, int* resultMatrix, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += matrix1[row * n + k] * matrix2[k * n + col];
        }
        resultMatrix[row * n + col] = sum;
    }
}

void printMatrix(int* matrix, int n) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			printf("[%d] ", matrix[i * n + j]);
		}
		printf("\n");
	}
}

// Fun��o para realizar opera��es com matrizes
void matrixOperations(int n) {
    printf("Valor de linhas e colunas: %d\n", n);

    auto born = std::chrono::high_resolution_clock::now();

    int* matrix1 = new int[n * n];
    int* matrix2 = new int[n * n];
    int* resultMatrix = new int[n * n];

    for (int i = 0; i < n * n; i++) {
        matrix1[i] = rand() % 100;
        matrix2[i] = rand() % 100;
        resultMatrix[i] = 0;
    }

    auto start = std::chrono::high_resolution_clock::now();
    long long bornChrono = std::chrono::duration_cast<std::chrono::microseconds>(start - born).count();
    printf("Tempo de criacao: %f \n", float(bornChrono) / 1000000);

    // Multiplica��o de matrizes em formato serial
    serialMatrixMultiplication(matrix1, matrix2, resultMatrix, n);

    auto resultChrono = std::chrono::high_resolution_clock::now() - start;
    long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(resultChrono).count();

    printf("Tempo para rodar (serial): %f \n", float(microseconds) / 1000000);

    // Aloca��o de mem�ria na GPU
    int* d_matrix, * d_auxiliarMatrix, * d_resultMatrix;
    hipMalloc(&d_matrix, n * n * sizeof(int));
    hipMalloc(&d_auxiliarMatrix, n * n * sizeof(int));
    hipMalloc(&d_resultMatrix, n * n * sizeof(int));

    // Copiar dados para a GPU
    hipMemcpy(d_matrix, matrix1, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_auxiliarMatrix, matrix2, n * n * sizeof(int), hipMemcpyHostToDevice);

    // Definir a quantidade de threads e blocos
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Multiplica��o de matrizes em paralelo
    start = std::chrono::high_resolution_clock::now();
    parallelMatrixMultiplication << <blocksPerGrid, threadsPerBlock >> > (d_matrix, d_auxiliarMatrix, d_resultMatrix, n);
    hipDeviceSynchronize();
    resultChrono = std::chrono::high_resolution_clock::now() - start;
    microseconds = std::chrono::duration_cast<std::chrono::microseconds>(resultChrono).count();
    printf("Tempo para rodar (paralelo): %f \n", float(microseconds) / 1000000);

    // Jogar resultado de volta para a CPU
    hipMemcpy(resultMatrix, d_resultMatrix, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Limpar mem�ria da GPU
    hipFree(d_matrix);
    hipFree(d_auxiliarMatrix);
    hipFree(d_resultMatrix);

    // Limpar mem�ria alocada
    delete[] matrix1;
    delete[] matrix2;
    delete[] resultMatrix;
}

int main() {
    int n;
    printf("Digite o valor de linhas e colunas: ");
    std::cin >> n;
    printf("\n");

    if (n == -1) {
        int sizes[] = { 10, 100, 500, 1000, 10000, 100000 };
        for (int size : sizes) {
            printf("Executando para matriz de tamanho %dx%d\n", size, size);
            matrixOperations(size);
            printf("\n");
        }
    }
    else {
        matrixOperations(n);
    }


    return 0;
}