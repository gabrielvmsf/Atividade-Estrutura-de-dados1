#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <>

const int TAMANHO_VETOR = 10000000;
const int NUM_BINS = 10;

__global__ void addKernel(double* vetor, int* histograma) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < TAMANHO_VETOR) {
        int bin = static_cast<int>(vetor[index]);
        if (bin < NUM_BINS) {
            atomicAdd(&histograma[bin], 1); 
        }
    }
}

void histogramaParalelo() {
    double* vetor = new double[TAMANHO_VETOR];
    double* d_vetor;
    int* histograma, * d_histograma;

    hipMalloc((void**)&d_histograma, NUM_BINS * sizeof(int));
    hipMalloc((void**)&d_vetor, TAMANHO_VETOR * sizeof(double));

    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < TAMANHO_VETOR; ++i) {
        vetor[i] = (double)rand() / RAND_MAX * 10;
    }

    hipMemcpy(d_vetor, vetor, TAMANHO_VETOR * sizeof(double), hipMemcpyHostToDevice);

    int histogramaInicial[NUM_BINS] = { 0 };
    hipMemcpy(d_histograma, histogramaInicial, NUM_BINS * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (TAMANHO_VETOR + blockSize - 1) / blockSize;
    addKernel << <numBlocks, blockSize >> > (d_vetor, d_histograma);

    histograma = new int[NUM_BINS];
    hipMemcpy(histograma, d_histograma, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);

    int sumParalelo = 0;
    for (int i = 0; i < NUM_BINS; ++i) {
        printf("Intervalo [%d, %d): %d\n", i, i + 1, histograma[i]);
        sumParalelo += histograma[i];
    }

    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

    printf("Somatorio: %d\n", sumParalelo);
    printf("Tempo de execucao: %.6f segundos\n", elapsed.count());

    delete[] vetor;
    delete[] histograma;
    hipFree(d_vetor);
    hipFree(d_histograma);
}

void histogramaSerial() {
    double* vetor = new double[TAMANHO_VETOR];
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < TAMANHO_VETOR; ++i) {
        vetor[i] = (double)rand() / RAND_MAX * 10;
    }

    int histograma[NUM_BINS] = { 0 };

    for (int i = 0; i < TAMANHO_VETOR; ++i) {
        int bin = static_cast<int>(vetor[i]);
        if (bin < NUM_BINS) {
            histograma[bin]++;
        }
    }
    int sumSerial = 0;

    for (int i = 0; i < NUM_BINS; ++i) {
        printf("Intervalo [%d, %d): %d\n", i, i + 1, histograma[i]);
        sumSerial += histograma[i];
    }

    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

    printf("Somatorio: %d\n", sumSerial);
    printf("Tempo de execucao: %.6f segundos\n", elapsed.count());
}

int main() {
    printf("Histograma Serial:\n");
    histogramaSerial();

    printf("Histograma Paralelo:\n");
    histogramaParalelo();

    return 0;
}