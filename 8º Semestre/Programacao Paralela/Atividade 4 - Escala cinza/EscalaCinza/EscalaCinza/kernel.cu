#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <chrono>
#include <vector>
#include <cstring>
#include <hip/hip_runtime.h>
#include <sstream> 



#ifdef _WIN32
#include <direct.h>
#define mkdir(path, mode) _mkdir(path)
#else
#include <sys/stat.h>
#endif

#define THREADS_PER_BLOCK 1024

// Estrutura para representar um pixel RGB
struct Pixel {
    unsigned char r, g, b;
};

// Função para criar diretório
void createDirectory(const std::string& path) {
#ifdef _WIN32
    _mkdir(path.c_str());
#else
    mkdir(path.c_str(), 0777);
#endif
}

// Função para ler o cabeçalho e dados da imagem BMP
bool readBMP(const std::string& filename, int& width, int& height, std::vector<Pixel>& pixels) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Erro ao abrir o arquivo BMP." << std::endl;
        return false;
    }

    // Ler cabeçalho BMP
    unsigned char header[54];
    file.read(reinterpret_cast<char*>(header), 54);

    // Validar formato BMP
    if (header[0] != 'B' || header[1] != 'M') {
        std::cerr << "Arquivo não é um BMP válido." << std::endl;
        return false;
    }

    // Extrair dimensões da imagem
    width = *(int*)&header[18];
    height = *(int*)&header[22];

    // Validar tamanho mínimo
    if (width < 500 || height < 500) {
        std::cerr << "Imagem muito pequena. Deve ter pelo menos 500x500 pixels." << std::endl;
        return false;
    }

    // Calcular tamanho do array de pixels
    int row_padded = (width * 3 + 3) & (~3);
    pixels.resize(width * height);

    // Ler pixels
    unsigned char* row = new unsigned char[row_padded];
    for (int i = 0; i < height; i++) {
        file.read(reinterpret_cast<char*>(row), row_padded);
        for (int j = 0; j < width; j++) {
            pixels[i * width + j] = { row[j * 3 + 2], row[j * 3 + 1], row[j * 3] };
        }
    }

    delete[] row;
    file.close();
    return true;
}

// Função para salvar a imagem em escala de cinza
bool saveBMP(const std::string& filename, int width, int height, const std::vector<unsigned char>& grayscale) {
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Erro ao salvar o arquivo BMP." << std::endl;
        return false;
    }

    // Cabeçalho BMP (54 bytes padrão)
    unsigned char header[54] = { 0 };
    header[0] = 'B';
    header[1] = 'M';
    *(int*)&header[2] = 54 + width * height; // Tamanho do arquivo
    header[10] = 54;                        // Offset para dados de pixel
    header[14] = 40;                        // Tamanho do cabeçalho DIB
    *(int*)&header[18] = width;
    *(int*)&header[22] = height;
    header[26] = 1;                         // Planos
    header[28] = 24;                        // Bits por pixel

    file.write(reinterpret_cast<char*>(header), 54);

    // Escrever pixels em escala de cinza
    int row_padded = (width * 3 + 3) & (~3);
    unsigned char* row = new unsigned char[row_padded];
    for (int i = 0; i < height; i++) {
        memset(row, 0, row_padded);
        for (int j = 0; j < width; j++) {
            unsigned char gray = grayscale[i * width + j];
            row[j * 3] = row[j * 3 + 1] = row[j * 3 + 2] = gray;
        }
        file.write(reinterpret_cast<char*>(row), row_padded);
    }

    delete[] row;
    file.close();
    return true;
}

// Função CUDA para conversão de escala de cinza
__global__ void convertToGrayscaleCUDA(Pixel* d_pixels, unsigned char* d_grayscale, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width * height) {
        Pixel p = d_pixels[idx];
        d_grayscale[idx] = static_cast<unsigned char>((p.r * 0.298 + p.g * 0.587 + p.b * 0.114) / 3.0);
    }
}

// Função serial para conversão
void convertToGrayscaleSerial(const std::vector<Pixel>& pixels, std::vector<unsigned char>& grayscale, int width, int height) {
    for (int i = 0; i < width * height; i++) {
        Pixel p = pixels[i];
        grayscale[i] = static_cast<unsigned char>((p.r * 0.298 + p.g * 0.587 + p.b * 0.114) / 3.0);
    }
}

// Função principal
int main() {
    const std::string inputFile = "input.bmp"; // Nome do arquivo de entrada
    const std::string outputDir = "output";   // Diretório base de saída

    // Criar diretório base
    createDirectory(outputDir);

    // Variáveis para armazenar informações da imagem
    int width, height;
    std::vector<Pixel> pixels;

    // Ler imagem BMP
    if (!readBMP(inputFile, width, height, pixels)) {
        return 1;
    }

    // Imprimir tamanho da imagem
    std::cout << "Dimensões da imagem: " << width << "x" << height << std::endl;

    // Vetor para armazenar a imagem em escala de cinza
    std::vector<unsigned char> grayscaleSerial(width * height);
    std::vector<unsigned char> grayscaleCUDA(width * height);

    // Repetições: 1, 10, 100, 1000
    std::vector<int> testCases = { 1, 10, 100, 1000 };
    std::ofstream resultFile(outputDir + "/performance_results.txt");

    // Iterar sobre os casos de teste
    for (int testCase : testCases) {
        std::cout << "\nExecutando para " << testCase << " imagens..." << std::endl;

        // Serial
        auto startSerial = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < testCase; i++) {
            convertToGrayscaleSerial(pixels, grayscaleSerial, width, height);
        }
        auto endSerial = std::chrono::high_resolution_clock::now();
        auto durationSerial = std::chrono::duration_cast<std::chrono::microseconds>(endSerial - startSerial);

        // Paralela (CUDA)
        Pixel* d_pixels;
        unsigned char* d_grayscale;
        hipMalloc(&d_pixels, pixels.size() * sizeof(Pixel));
        hipMalloc(&d_grayscale, grayscaleCUDA.size() * sizeof(unsigned char));
        hipMemcpy(d_pixels, pixels.data(), pixels.size() * sizeof(Pixel), hipMemcpyHostToDevice);

        auto startCUDA = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < testCase; i++) {
            int totalPixels = width * height;
            int blocks = (totalPixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            convertToGrayscaleCUDA << <blocks, THREADS_PER_BLOCK >> > (d_pixels, d_grayscale, width, height);
            hipMemcpy(grayscaleCUDA.data(), d_grayscale, grayscaleCUDA.size() * sizeof(unsigned char), hipMemcpyDeviceToHost);
        }
        auto endCUDA = std::chrono::high_resolution_clock::now();
        auto durationCUDA = std::chrono::duration_cast<std::chrono::microseconds>(endCUDA - startCUDA);

        hipFree(d_pixels);
        hipFree(d_grayscale);

        // Salvar resultados
        std::string serialOutputDir = outputDir + "/serial_case_" + std::to_string(testCase);
        std::string cudaOutputDir = outputDir + "/cuda_case_" + std::to_string(testCase);
        createDirectory(serialOutputDir);
        createDirectory(cudaOutputDir);

        // Salvar cada imagem processada
        for (int i = 0; i < testCase; i++) {
            saveBMP(serialOutputDir + "/output_" + std::to_string(i + 1) + ".bmp", width, height, grayscaleSerial);
            saveBMP(cudaOutputDir + "/output_" + std::to_string(i + 1) + ".bmp", width, height, grayscaleCUDA);
        }

        // Escrever tempos no arquivo
        resultFile << "Test Case: " << testCase << " imagens\n";
        resultFile << "Serial: " << durationSerial.count() << " µs\n";
        resultFile << "CUDA: " << durationCUDA.count() << " µs\n";
        resultFile << "---------------------------------\n";

        // Exibir resultados no console
        std::cout << "Serial: " << durationSerial.count() << " microsegundos" << std::endl;
        std::cout << "CUDA: " << durationCUDA.count() << " microsegundos" << std::endl;
    }

    resultFile.close();
    std::cout << "\nResultados salvos em: " << outputDir + "/performance_results.txt" << std::endl;

    return 0;
}

template <typename T>
std::string numberToString(const T& value) {
    std::ostringstream oss;
    oss << value;
    return oss.str();
}