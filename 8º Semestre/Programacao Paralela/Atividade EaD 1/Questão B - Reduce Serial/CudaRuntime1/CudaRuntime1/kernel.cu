﻿#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

// Função de Reduce Serial na CPU
float reduceSerial(const std::vector<float>& data) {
    float result = 0;
    for (size_t i = 0; i < data.size(); ++i) {
        result += data[i];
    }
    return result;
}

// Estrutura para armazenar informações de desempenho
struct PerformanceMetrics {
    float executionTime; // Tempo de execução em milissegundos
    size_t workAmount;    // Quantidade de trabalho (número de elementos)
    int steps;            // Número de passos (neste caso, pode ser o tamanho do vetor)
};

// Função para medir tempo de execução e coletar métricas
PerformanceMetrics measureTime(const std::vector<float>& data) {
    PerformanceMetrics metrics;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    float result = reduceSerial(data); // Executa o Reduce Serial
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&metrics.executionTime, start, stop);
    metrics.workAmount = data.size(); // Quantidade de trabalho é o tamanho do vetor
    metrics.steps = metrics.workAmount; // Número de passos pode ser o mesmo que o tamanho do vetor
    std::cout << "Resultado da soma: " << result << "\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return metrics;
}

int main() {
    std::srand(static_cast<unsigned>(std::time(0)));

    // Diferentes tamanhos de arrays para teste
    std::vector<int> sizes = { 100, 1000, 10000, 100000, 1000000, 10000000 };

    for (int size : sizes) {
        std::vector<float> data(size);

        // Preenchendo o array com valores aleatórios
        for (int i = 0; i < size; ++i) {
            data[i] = static_cast<float>(std::rand()) / RAND_MAX;
        }

        // Medindo o tempo de execução e coletando métricas
        PerformanceMetrics metrics = measureTime(data);
        std::cout << "Tamanho do array: " << size
            << " -> Tempo de execução: " << metrics.executionTime << " ms"
            << ", Quantidade de trabalho: " << metrics.workAmount
            << ", Número de passos: " << metrics.steps << "\n";
    }

    return 0;
}