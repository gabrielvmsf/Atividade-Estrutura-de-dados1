#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

// Kernel para a Fase Up-Sweep (Redução)
__global__ void upSweepKernel(float* d_data, int n, int step) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int distance = 1 << step; // Distância para o passo atual

    if (idx < n && idx % (distance * 2) == 0 && idx + distance < n) {
        d_data[idx + distance * 2 - 1] += d_data[idx + distance - 1];
    }
}

// Kernel para a Fase Down-Sweep (Distribuição)
__global__ void downSweepKernel(float* d_data, int n, int step) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int distance = 1 << step;

    if (idx < n && idx % (distance * 2) == 0 && idx + distance < n) {
        float temp = d_data[idx + distance - 1];
        d_data[idx + distance - 1] = d_data[idx + distance * 2 - 1];
        d_data[idx + distance * 2 - 1] += temp;
    }
}

// Função principal que executa o algoritmo de Blelloch
float blellochScan(const std::vector<float>& input) {
    int n = input.size();
    float* d_data;

    // Aloca memória na GPU e copia o vetor inicial para a GPU
    hipMalloc(&d_data, n * sizeof(float));
    hipMemcpy(d_data, input.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int steps = std::ceil(std::log2(n)); // Número de etapas
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Fase Up-Sweep (Redução)
    for (int step = 0; step < steps; ++step) {
        upSweepKernel << <gridSize, blockSize >> > (d_data, n, step);
        hipDeviceSynchronize();  // Certifique-se de que a etapa esteja concluída
    }

    // Inicializa o último valor com 0 para a Fase Down-Sweep
    hipMemset(&d_data[n - 1], 0, sizeof(float));

    // Fase Down-Sweep (Distribuição)
    for (int step = steps - 1; step >= 0; --step) {
        downSweepKernel << <gridSize, blockSize >> > (d_data, n, step);
        hipDeviceSynchronize();  // Certifique-se de que a etapa esteja concluída
    }

    // Registra o tempo de execução
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copia o resultado de volta para a CPU
    std::vector<float> result(n);
    hipMemcpy(result.data(), d_data, n * sizeof(float), hipMemcpyDeviceToHost);

    // Libera a memória da GPU
    hipFree(d_data);

    std::cout << "Último valor do vetor resultado: " << result.back() << "\n";
    return time;
}

int main() {
    std::vector<int> sizes = { 100, 1000, 10000, 100000, 1000000, 10000000 };

    for (int size : sizes) {
        std::vector<float> input(size, 1.0f); // Inicializa o array com valores 1.0 para simplificar a verificação
        int steps = std::ceil(std::log2(size)); // Número de etapas
        int work = size * steps; // Quantidade de trabalho total realizada

        std::cout << "Tamanho do array: " << size << "\n";
        float time = blellochScan(input);
        std::cout << "Tempo de execução: " << time << " ms\n";
        std::cout << "Quantidade de trabalho: " << work << " operações\n";
        std::cout << "Número de passos: " << steps << "\n\n";
    }

    return 0;
}
