#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

// Função kernel para realizar uma etapa de Hillis-Steele
__global__ void hillisSteeleKernel(float* d_input, float* d_output, int n, int step) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = 1 << step; // Calcula 2^step

    if (idx < n) {
        if (idx >= offset) {
            d_output[idx] = d_input[idx] + d_input[idx - offset];
        }
        else {
            d_output[idx] = d_input[idx];
        }
    }
}

// Estrutura para armazenar informações de desempenho
struct PerformanceMetrics {
    float executionTime; // Tempo de execução em milissegundos
    size_t workAmount;    // Quantidade de trabalho (número de operações)
    int steps;            // Número de passos (etapas)
};

// Função que executa o algoritmo Hillis-Steele e coleta métricas
PerformanceMetrics hillisSteele(const std::vector<float>& input) {
    int n = input.size();
    float* d_input;
    float* d_output;

    // Aloca memória na GPU
    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    // Copia dados para a GPU
    hipMemcpy(d_input, input.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int steps = std::ceil(std::log2(n)); // Número de etapas
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Loop pelas etapas de Hillis-Steele
    for (int s = 0; s < steps; ++s) {
        hillisSteeleKernel << <gridSize, blockSize >> > (d_input, d_output, n, s);
        hipMemcpy(d_input, d_output, n * sizeof(float), hipMemcpyDeviceToDevice); // Atualiza o vetor de entrada
    }

    // Registra o tempo de execução
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copia o resultado de volta para a CPU
    std::vector<float> output(n);
    hipMemcpy(output.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Libera a memória da GPU
    hipFree(d_input);
    hipFree(d_output);

    std::cout << "Resultado final: " << output.back() << "\n";

    // Coleta as métricas de desempenho
    PerformanceMetrics metrics;
    metrics.executionTime = time;
    metrics.workAmount = n * steps; // Quantidade de trabalho (número de operações)
    metrics.steps = steps; // Número de etapas

    return metrics;
}

int main() {
    std::vector<int> sizes = { 100, 1000, 10000, 100000, 1000000, 10000000 };

    for (int size : sizes) {
        std::vector<float> input(size, 1.0f); // Inicializa o array com valores 1.0 para simplificar a verificação

        std::cout << "Tamanho do array: " << size << "\n";
        PerformanceMetrics metrics = hillisSteele(input);
        std::cout << "Tempo de execução: " << metrics.executionTime << " ms\n";
        std::cout << "Quantidade de trabalho: " << metrics.workAmount << " operações\n";
        std::cout << "Número de passos: " << metrics.steps << "\n\n";
    }

    return 0;
}