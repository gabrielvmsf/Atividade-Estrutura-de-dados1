#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

using namespace std;
using namespace std::chrono;

// Funcao de multiplicacao serial
void serialMatrixVectorMultiplication(const vector<float>& matrix, const vector<float>& vec, vector<float>& result, int m, int n) {
    int denseCount = 0;
    for (int i = 0; i < m; i++) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            if (matrix[IDX2C(i, j, m)] != 0) {  // Ignora valores esparsos
                sum += matrix[IDX2C(i, j, m)] * vec[j];
                denseCount++;
            }
        }
        result[i] = sum;
    }
    cout << "Dados densos (nao-zero) na matriz: " << denseCount << endl;
}

// Funcao Kernel para multiplicacao paralela
__global__ void sparseMatrixVectorMultiplication(const float* matrix, const float* vec, float* result, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            float val = matrix[IDX2C(row, j, m)];
            if (val != 0) {  // Ignora valores esparsos
                sum += val * vec[j];
            }
        }
        result[row] = sum;
    }
}

void runTest(int m, int n) {
    // Inicializa a matriz e o vetor com valores esparsos
    vector<float> matrix(m * n, 0);
    vector<float> vec(n, 1.0f);
    vector<float> result_serial(m, 0.0f);
    vector<float> result_parallel(m, 0.0f);

    // Preenche a matriz com alguns valores densos
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (rand() % 4 == 0) {  // Coloca um valor diferente de zero em 25% das posicoes
                matrix[IDX2C(i, j, m)] = static_cast<float>(rand() % 10 + 1);
            }
        }
    }

    // Multiplicacao Serial
    auto start = high_resolution_clock::now();
    serialMatrixVectorMultiplication(matrix, vec, result_serial, m, n);
    auto end = high_resolution_clock::now();
    double serialTime = duration_cast<microseconds>(end - start).count();
    cout << "\n--- Multiplicacao Serial ---" << endl;
    cout << "Tempo de execucao serial: " << serialTime << " microsegundos" << endl;

    // Preparacao para multiplicacao paralela
    float* d_matrix, * d_vec, * d_result;
    hipMalloc((void**)&d_matrix, m * n * sizeof(float));
    hipMalloc((void**)&d_vec, n * sizeof(float));
    hipMalloc((void**)&d_result, m * sizeof(float));

    hipMemcpy(d_matrix, matrix.data(), m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec.data(), n * sizeof(float), hipMemcpyHostToDevice);

    // Multiplicacao Paralela
    int blockSize = 256;
    int numBlocks = (m + blockSize - 1) / blockSize;

    start = high_resolution_clock::now();
    sparseMatrixVectorMultiplication << <numBlocks, blockSize >> > (d_matrix, d_vec, d_result, m, n);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();

    double parallelTime = duration_cast<microseconds>(end - start).count();
    cout << "--- Multiplicacao Paralela ---" << endl;
    cout << "Tempo de execucao paralela: " << parallelTime << " microsegundos" << endl;

    // Copia resultado de volta
    hipMemcpy(result_parallel.data(), d_result, m * sizeof(float), hipMemcpyDeviceToHost);

    // Libera memoria
    hipFree(d_matrix);
    hipFree(d_vec);
    hipFree(d_result);

    // Validacao e comparacao de desempenho
    bool isCorrect = true;
    for (int i = 0; i < m; i++) {
        if (abs(result_serial[i] - result_parallel[i]) > 1e-5) {
            isCorrect = false;
            break;
        }
    }
    cout << "--- Resultados ---" << endl;
    cout << "Resultados corretos: " << (isCorrect ? "Sim" : "Nao") << endl;
    cout << "Aceleracao obtida: " << serialTime / parallelTime << "x" << endl;
}

int main() {
    srand(time(0));

    cout << "\n--- Teste de Multiplicacao Matriz-Vetor ---" << endl;

    cout << "\nTeste 1: 100x100" << endl;
    runTest(100, 100);

    cout << "\nTeste 2: 1000x100" << endl;
    runTest(1000, 100);

    cout << "\nTeste 3: 10000x100" << endl;
    runTest(10000, 100);

    cout << "\nTeste 4: 100000x100" << endl;
    runTest(100000, 100);

    cout << "\nTeste 5: 1000000x100" << endl;
    runTest(1000000, 100);

    cout << "\nTeste 6: 10000000x100" << endl;
    runTest(10000000, 100);

    return 0;
}
